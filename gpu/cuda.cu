// Instalación NVIDIA toolkit: https://docs.nvidia.com/cuda/cuda-installation-guide-linux/
// Compilación:
// $ nvcc cuda_add.cu -o cuda_add

// Requiere hardware GPU NVIDIA
// Verificar en su sistema, ejemplo Ubuntu 22
// $ nvidia-detector

#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addValue(int* arr, int val, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < n) {
        arr[idx] += val;
    }
}

int main() {
    int n = 5;
    int size = n * sizeof(int);
    int h_data[5] = {1, 2, 3, 4, 5};
    int *d_data;

    // Reservar memoria en la GPU
    hipMalloc((void**)&d_data, size);
    // Copiar datos desde la CPU (host) a la GPU (device)
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    // Lanzar kernel: 1 bloque, n hilos por bloque
    addValue<<<1, n>>>(d_data, 10, n);
    // Esperar a que la GPU termine
    hipDeviceSynchronize();

    // Copiar resultados de la GPU al host
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    for(int i=0; i<n; i++){
        std::cout << "Pos " << i << " = " << h_data[i] << std::endl;
    }

    hipFree(d_data);
    return 0;
}
